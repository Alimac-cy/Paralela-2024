// Written by Barry Wilkinson, UNC - Charlotte.Pi.cu December 22,2010.
// Derived somewhat from code developed by Patrick Rogers, UNC-C
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#define TRIALS_PER_THREAD 4096
#define BLOCKS 256
#define THREADS 256
#define PI 3.1415926535 // known value of pi
__global__ void gpu_monte_carlo(float *estimate, hiprandState *states)
{
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int points_in_circle = 0;
    float x, y;
    hiprand_init(1234, tid, 0, &states[tid]);
    hiprandState localState = states[tid];

    for (int i = 0; i < TRIALS_PER_THREAD; i++)
    {
        // Generate random numbers
        x = hiprand_uniform(&localState) ;
        y = hiprand_uniform(&localState) ;

        points_in_circle += (x * x + y * y <= 1.0f); // Count if (x, y) is in the circle.
    }

    states[tid] = localState;

    estimate[tid] = 4.0f * points_in_circle / (float)TRIALS_PER_THREAD; // Return estimate of pi
}

float host_monte_carlo(long trials)
{
    float x, y;
    long points_in_circle;
    for (long i = 0; i < trials; i++)
    {
        x = rand() / (float)RAND_MAX;
        y = rand() / (float)RAND_MAX;
        points_in_circle += (x * x + y * y <= 1.0f);
    }
    return 4.0f * points_in_circle / trials;
}
int main(int argc, char *argv[])
{
    clock_t start, stop;
    float host[BLOCKS * THREADS];
    float *dev;
    hiprandState *devStates;
    printf("# of trials per thread = %d, # of blocks = %d, # of threads / block = % d.\n ", TRIALS_PER_THREAD, BLOCKS, THREADS);
    start = clock();
    // TODO: use cudaMalloc para pedir un bloque de memoria de tamaño BLOCKS *THREADS * sizeof(float).Guárdelo en dev.
    // TODO: use cudaMalloc para pedir un bloque de memoria de tamaño THREADS *BLOCKS *sizeof(curandState).Será usado para guardar el estado del generador de números aleatorios.Guardelo en devStates.gpu_monte_carlo<<<BLOCKS, THREADS>>>(dev, devStates);
    hipMalloc((void **)&dev, BLOCKS * THREADS * sizeof(float));
    hipMalloc((void **)&devStates, BLOCKS * THREADS * sizeof(hiprandState));
    // TODO: Use cudaMemcpy para copiar los datos de dev a host.
    gpu_monte_carlo<<<BLOCKS, THREADS>>>(dev, devStates);
    hipMemcpy(host, dev, BLOCKS * THREADS * sizeof(float), hipMemcpyDeviceToHost);
    float pi_gpu;
    for (int i = 0; i < BLOCKS * THREADS; i++)
    {
        pi_gpu += host[i];
    }
    pi_gpu /= (BLOCKS * THREADS);
    stop = clock();
    printf("GPU pi calculated in %f s.\n", (stop -
                                            start) /
                                               (float)CLOCKS_PER_SEC);
    start = clock();
    float pi_cpu = host_monte_carlo(BLOCKS * THREADS *
                                    TRIALS_PER_THREAD);
    stop = clock();
    printf("CPU pi calculated in %f s.\n", (stop -
                                            start) /
                                               (float)CLOCKS_PER_SEC);
    printf("CUDA estimate of PI = %f [error of %f]\n", pi_gpu,
           pi_gpu - PI);
    printf("CPU estimate of PI = %f [error of %f]\n", pi_cpu,
           pi_cpu - PI);
    hipFree(dev);
    hipFree(devStates);
    return 0;
}
